
#include <hip/hip_runtime.h>
#include <iostream>
#include <bit>
#include <bitset>
#include <cstdint>

#define MAX_NBR_FACETS 252
#define NBR_RIDGES 2688
#define NBR_X0 2143588
#define SIZE_X 198414832
#define RESULT_SIZE (1ul<<32)

using namespace std;

__shared__ int r[NBR_RIDGES];
__device__  unsigned int ai[4][210];
__device__ int mi[4][11][210];
__device__ unsigned int X[SIZE_X];
__device__ __managed__ long out[RESULT_SIZE];
__device__ __managed__ int n_out = 0;
struct StructX0 {
    unsigned long X0;
    unsigned int precalc[27];
};

const unsigned long nbrX0 = NBR_X0;
StructX0 listX0[nbrX0];


__global__ void kernel(StructX0 structX0[]) {
    unsigned int a[4];
    unsigned int precalc_a = structX0[blockIdx.x].precalc[threadIdx.x / 8];
    unsigned long X0 = structX0[blockIdx.x].X0;
    for (int k = 0; k < 4; k++) {
        a[k] = ai[k][threadIdx.x] | ((precalc_a >> (4 * (threadIdx.x % 8) + k)) & 1u) << 31;
    }
    int m[4][11];
    for (int k = 0; k < 4; k++) {
        for (int l = 0; l < 11; l++) {
            m[k][l] = mi[k][l][threadIdx.x];
        }
    }
    bool Ax[4];
    bool stop;
    for (unsigned int x: X) {
        for (int j = 0; j < 4; j++) {
            Ax[j] = __popc(a[j] & x) & 1;
        }
        int count = 0;
        for (bool j: Ax) {
            count += __syncthreads_count(j);
        }
        if (count > MAX_NBR_FACETS) continue;
        for (bool j: Ax) {
            if (j) {
                for (int k = 0; k < 11; k++) {
                    if (atomicAdd(r + m[j][k], 1) >= 4) {
                        stop = true;
                    }
                }
            }
        }
        if (__syncthreads_or(stop)) continue;
        if (threadIdx.x == 0) {
            out[atomicAdd(&n_out, 1)] = X0 | x;
        }
    }

}

void increment_vect(unsigned int vect[], const unsigned int ref[], const int size) {
    vect[0] = (vect[0] + 1) % ref[0];
    int k = 0;
    while (vect[k] == 0 and k < size - 1) {
        k += 1;
        vect[k] = (vect[k] + 1) % ref[k];
    }
}
void calculs_GPU(unsigned long A[], unsigned int M[]) {
    unsigned int list_groups[20] = {1, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 4, 3, 1, 1, 1, 1};
    unsigned int list_shifts[20];
    unsigned int list_ref[19];
    list_shifts[19] = 0;
    for (int k = 18; k > -1; k--) {
        list_shifts[k] = list_groups[k + 1] + list_shifts[k + 1];
    }
    unsigned long list_elementary[19][11];
    for (int i = 0; i < 19; i++) {
        int position = 0;
        for (int j = 0; j < (1ul << (list_groups[i + 1])); j++) {
            if (__popcount(j) <= 2) {
                unsigned long jl = j;
                list_elementary[i][position] = (jl << list_shifts[i + 1]);
                position += 1;
            }
        }
        list_ref[i] = position;
    }
    int size = 8;
    unsigned int vect[size];
    for (int k = 0; k < size; k++) {
        vect[k] = 0;
    }
    unsigned long index = 0;
    while (index < nbrX0) {
        unsigned long x = (1ul << 63);
        for (int i = 0; i < size; i++) {
            x |= list_elementary[i][vect[i]];
        }
        listX0[index].X0 = x;
        increment_vect(vect, list_ref, size);
        if (index % 1000 == 0) {
            for (unsigned int l: vect) {
                cout << l << ',';
            }
            cout << '\n';
        }
        index++;
    }

    for(int k=0;k<nbrX0;k++){
        for (int i =0;i<1;i++){

        }
    }
    // Enumérer les X1 31
    // lancer le 210 000 000 précalc correspondant à X0 blocks
    kernel<<<1, 210>>>(listX0);
    for (int i = 0; i < n_out; i++) {
        cout << out[i];
        // écrire dans le fichier texte out[k] (printf)
    }
}